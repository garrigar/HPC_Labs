
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

#define GPU_ID 0

hipblasHandle_t HANDLE = NULL;
float *dev_A = NULL, *dev_B = NULL, *dev_C = NULL;
int LAST_M = -1, LAST_N = -1, LAST_K = -1;


static void handleError(hipError_t error, const char *file, int line) {
    if (error != hipSuccess)
    {
        fprintf(stderr, "Error: %s:%d, ", file, line);
        fprintf(stderr, "code: %d, reason: %s\n", error, hipGetErrorString(error));
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) ( handleError( err, __FILE__, __LINE__ ) )


void gpu_blas_matmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    int lda = m;
    int ldb = k;
    int ldc = m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    if (HANDLE == NULL){
        hipblasCreate(&HANDLE); 
    }
    
    // C = alpha * op(A)op(B) + beta * C
    // ld? - leading dimensions of the matrices
    hipblasSgemm(HANDLE, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc); 
    
    // cublasDestroy(HANDLE); 
}

// float* cudaMallocMatrix(const int m, const int n){
//     float *dev_matrix;
//     HANDLE_ERROR(cudaMalloc(&dev_matrix, m * n * sizeof(float)));
//     return dev_matrix;
// }

// returns: time in ms
float gpu_matmul(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    hipSetDevice(GPU_ID);

    size_t size_A = m * k * sizeof(float);
    size_t size_B = k * n * sizeof(float);
    size_t size_C = m * n * sizeof(float);

    // reallocating memory only if new dims are different
    if ((m != LAST_M) || (k != LAST_K) || (n != LAST_N)) {
        // freeing memory if it is not "empty"
        // if (dev_A != NULL || dev_B != NULL || dev_C != NULL) {
            HANDLE_ERROR(hipFree(dev_A));
            HANDLE_ERROR(hipFree(dev_B));
            HANDLE_ERROR(hipFree(dev_C));
        // }
        LAST_M = m;
        LAST_K = k;
        LAST_N = n;
        HANDLE_ERROR(hipMalloc(&dev_A, size_A));
        HANDLE_ERROR(hipMalloc(&dev_B, size_B));
        HANDLE_ERROR(hipMalloc(&dev_C, size_C));
    }


    HANDLE_ERROR(hipMemcpy(dev_A, A, size_A, hipMemcpyHostToDevice)); 
    HANDLE_ERROR(hipMemcpy(dev_B, B, size_B, hipMemcpyHostToDevice));

    hipEvent_t start, stop; 
    float time = 0.0;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));
    
    gpu_blas_matmul(dev_A, dev_B, dev_C, m, k, n); 
    
    HANDLE_ERROR(hipDeviceSynchronize());
    HANDLE_ERROR(hipEventRecord(stop, 0));
        
    HANDLE_ERROR(hipMemcpy(C, dev_C, size_C, hipMemcpyDeviceToHost));
    
    HANDLE_ERROR(hipEventElapsedTime(&time, start, stop));

    return time; // ms

}
